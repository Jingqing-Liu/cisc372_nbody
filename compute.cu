#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <hip/hip_runtime.h>
	
//first compute the pairwise accelerations.  Effect is on the first argument.
__global__ void compute_Pairwise_Accelerations(vector3 *hPos, double *mass, vector3 *accels, int numEntities) {

	__shared__ double share_mass[16];
	__shared__ vector3 share_hPos[16];

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < numEntities && threadIdx.x == 0) {
		share_mass[threadIdx.y] = mass[i];
		share_hPos[threadIdx.y][0] = hPos[i][0];
		share_hPos[threadIdx.y][1] = hPos[i][1];
		share_hPos[threadIdx.y][2] = hPos[i][2];
	}

	__syncthreads();

	if (i < numEntities && j < numEntities) {
		if (i == j) {
			FILL_VECTOR(accels[i * numEntities + j], 0, 0, 0);
		} else {
			vector3 distance;
			for (int k = 0; k < 3; k++) {
				distance[k] = hPos[i][k] - hPos[j][k];
			}
			double magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];
			double magnitude = sqrt(magnitude_sq);
			double accelmag = -1 * GRAV_CONSTANT * mass[j] / magnitude_sq;
			FILL_VECTOR(accels[i * numEntities + j], accelmag * distance[0] / magnitude, accelmag * distance[1] / magnitude, accelmag * distance[2] / magnitude);
		}
	} 
}

//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
__global__ void update_velocity_and_position(vector3* hPos, vector3* hVel, vector3* accels, int numEntities, double interval) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < numEntities) {
	//compute the new velocity based on the acceleration and time interval
	//compute the new position based on the velocity and time interval
		for (int k = 0; k < 3; k++){
			hVel[i][k] += accel_sum[k] * interval;
			hPos[i][k] = hVel[i][k] * interval;
		}
	}
}

__global__ void sum(vector3* accels, vector3* sum_accels, int numEntities) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < numEntities) {
		vector3 sum={0, 0, 0};
		for (int j = 0; j < numEntities; j++){
			for (int k = 0;k < 3; k++) {
				accel_sum[k] += accels[i * numEntities + j][k];
			}
		}
		sun_accels[i][0] = sum[0];
		sun_accels[i][1] = sum[1];
		sun_accels[i][2] = sum[2];
	}
}


//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){

	vector3 *device_hPos, *device_hVel, *device_accels;
	double *device_mass;

	hipMalloc((void**)&device_hPos, sizeof(vector3)*NUMENTITIES);
	hipMalloc((void**)&device_hVel, sizeof(vector3)*NUMENTITIES);
	hipMalloc((void**)&device_mass, sizeof(double)*NUMENTITIES);
	hipMalloc((void**)&device_accels, sizeof(vector3)*NUMENTITIES*NUMENTITIES);

	hipMemcpy(device_hPos, hPos, sizeof(vector3)*NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(device_hVel, hVel, sizeof(vector3)*NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(device_mass, mass, sizeof(double)*NUMENTITIES, hipMemcpyHostToDevice);

	dim3 blockDim(16, 16);
	dim3 gridDim((NUMENTITIES + blockDim.x - 1) / blockDim.x, (NUMENTITIES + blockDim.y - 1) / blockDim.y);

	compute_Pairwise_Accelerations<<<gridDim, blockDim>>>(device_hPos, device_mass, device_accels, NUMENTITIES);

	hipDeviceSynchronize();

	sum_and_update_velocity_and_position<<<gridDim.x, blockDim.x>>>(device_hPos, device_hVel, device_accels, NUMENTITIES, INTERVAL);

	hipMemcpy(hPos, device_hPos, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hVel, device_hVel, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);

	hipFree(device_hPos);
	hipFree(device_hVel);
	hipFree(device_mass);
	hipFree(device_accels);
}
